#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <vector>

struct Point {
    size_t x;
    size_t y;
    uint8_t mask;
};

__global__ void mtla_matmul_kernel(
    const __hip_bfloat16 *a,
    const __hip_bfloat16 *b,
    __hip_bfloat16 *out,
    size_t col_num,
    size_t row_num,
    const Point *points,
    size_t points_len
) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= points_len) {
        return;
    }

    size_t point_idx = tid % points_len;
    size_t batch_idx = tid / points_len;

    const __hip_bfloat16 *a_offset = a + batch_idx * row_num * col_num;
    const __hip_bfloat16 *b_offset = b + batch_idx * row_num * col_num;
    __hip_bfloat16 *out_offset = out + batch_idx * row_num * col_num;

    const Point &point = points[point_idx];
    size_t x = point.x;
    size_t y = point.y;
    uint8_t mask = point.mask;

    __hip_bfloat16 *out_point = out_offset + y * row_num + x;
    const __hip_bfloat16 *row = a_offset + y * col_num;

    __hip_bfloat16 out_point_tmp = __float2bfloat16(0.0f);
    for (size_t i = 0; i < col_num; i++) {
        __hip_bfloat16 a_v = *(row + i);

        if (mask == 1) {
            __hip_bfloat16 b_v = b_offset[x * col_num + i];
            out_point_tmp += a_v * b_v;
        } else if (mask == 2) {
            __hip_bfloat16 b_v = b_offset[(x - 1) * col_num + i] +
                                b_offset[x * col_num + i];
            out_point_tmp += a_v * b_v;
        } else if (mask == 4) {
            __hip_bfloat16 b_v = b_offset[(x - 3) * col_num + i] +
                                b_offset[(x - 2) * col_num + i] +
                                b_offset[(x - 1) * col_num + i] +
                                b_offset[x * col_num + i];
            out_point_tmp += a_v * b_v;
        }
    }
    *out_point = out_point_tmp;
}

uint8_t find_mask(size_t x, size_t y, size_t window) {
    if (x > y) {
        return 0;
    }

    size_t x_num = x + 1;
    size_t y_num = y + 1;
    size_t new_window = window;

    if (y_num % 2 != 0) {
        new_window += 1;
    }

    if (y_num <= new_window) {
        return 1;
    }

    size_t not_in_window = y_num - new_window;

    if (not_in_window < x_num) {
        return 1;
    }

    if (x_num % 2 != 0) {
        return 0;
    }

    size_t not_in_vwindow = not_in_window - std::min(window, not_in_window);

    if (not_in_vwindow < x_num) {
        return 2;
    }

    if (not_in_vwindow < 4) {
        return 2;
    }

    if (x_num % 4 == 0) {
        return 4;
    } else {
        if (x_num == not_in_vwindow) {
            return 2;
        }
        return 0;
    }
}

std::vector<Point> gen_point_list(size_t mat_rows, size_t window) {
    std::vector<Point> threads;

    for (size_t y = 0; y < mat_rows; ++y) {
        for (size_t x = 0; x < mat_rows; ++x) {
            if (x > y) {
                break;
            }

            uint8_t mask = find_mask(x, y, window);

            if (mask != 0) {
                threads.push_back(Point{x, y, mask});
            }
        }
    }

    return threads;
}

void mtla_matmul(
    size_t a,
    size_t b,
    size_t out,
    size_t col_num,
    size_t row_num,
    size_t batch_size,
    size_t window,
    uint64_t stream_int
) {
    hipStream_t stream = reinterpret_cast<hipStream_t>(stream_int);

    std::vector<Point> points = gen_point_list(row_num, window);
    size_t points_len = points.size() * batch_size;

    size_t thread = 256;
    size_t block = points_len / thread;
    if (points_len % thread != 0) {
        block += 1;
    }

    Point* d_points = nullptr;
    hipMallocAsync(&d_points, points.size() * sizeof(Point), stream);
    hipMemcpyAsync(d_points, points.data(), points.size() * sizeof(Point), hipMemcpyHostToDevice, stream);
    mtla_matmul_kernel<<<block, thread, 0, stream>>>(
        reinterpret_cast<const __hip_bfloat16*>(a),
        reinterpret_cast<const __hip_bfloat16*>(b),
        reinterpret_cast<__hip_bfloat16*>(out),
        col_num,
        row_num,
        d_points,
        points.size()
    );
    hipFreeAsync(d_points, stream);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("mtla_matmul", &mtla_matmul, "mtla_matmul");
}